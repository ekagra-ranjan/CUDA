
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define BLOCKSIZE 256

/**********/
/* iDivUp */
/**********/
int iDivUp(int a, int b){ return ((a % b) != 0) ? (a / b + 1) : (a / b); }

/***********************/
/* CUDA ERROR CHECKING */
/***********************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line); 
        if (abort) exit(code);
    }
}

/********************************************************/
/* KERNEL FUNCTION FOR TESTING RANDOM NUMBER GENERATION */
/********************************************************/
__global__ void testrand1(unsigned long seed, float *a, int N){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprandState state;
    if (idx < N) {
        hiprand_init(seed, idx+2, 0, &state);
        a[idx] = hiprand_uniform(&state);
    }
}

/********/
/* MAIN */
/********/
int main() {

    const int N = 10;

    float *h_a  = (float*)malloc(N*sizeof(float));
    float *d_a; gpuErrchk(hipMalloc((void**)&d_a, N*sizeof(float)));

    testrand1<<<iDivUp(N, BLOCKSIZE), BLOCKSIZE>>>(1234, d_a, N);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    gpuErrchk(hipMemcpy(h_a, d_a, N*sizeof(float), hipMemcpyDeviceToHost));

    for (int i=0; i<N; i++) printf("%i %f\n", i, h_a[i]);

    // getchar();
}

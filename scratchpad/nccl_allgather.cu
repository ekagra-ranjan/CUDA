#include <nccl.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

#define NUM_GPUS 2  // Change this based on available GPUs

#define CHECK_CUDA(call)                                                      \
    do {                                                                      \
        hipError_t err = call;                                               \
        if (err != hipSuccess) {                                             \
            std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE);                                               \
        }                                                                     \
    } while (0)

#define CHECK_NCCL(call)                                                      \
    do {                                                                      \
        ncclResult_t err = call;                                              \
        if (err != ncclSuccess) {                                             \
            std::cerr << "NCCL Error: " << ncclGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE);                                               \
        }                                                                     \
    } while (0)

void all_gather_example() {
    int numDevices = NUM_GPUS;
    std::vector<int> devices(numDevices);
    std::vector<hipStream_t> streams(numDevices);
    std::vector<ncclComm_t> comms(numDevices);
    std::vector<float*> d_sendBuf(numDevices);
    std::vector<float*> d_recvBuf(numDevices);

    size_t dataSize = 4; // Each GPU sends 4 floats
    size_t totalDataSize = dataSize * numDevices;

    // Initialize devices
    for (int i = 0; i < numDevices; i++) {
        devices[i] = i;
        CHECK_CUDA(hipSetDevice(i));

        // Allocate memory for sending and receiving data
        CHECK_CUDA(hipMalloc(&d_sendBuf[i], dataSize * sizeof(float)));
        CHECK_CUDA(hipMalloc(&d_recvBuf[i], totalDataSize * sizeof(float)));

        // Initialize stream
        CHECK_CUDA(hipStreamCreate(&streams[i]));

        // Fill send buffer with unique values for each GPU
        std::vector<float> h_sendBuf(dataSize, i + 1);
        CHECK_CUDA(hipMemcpy(d_sendBuf[i], h_sendBuf.data(), dataSize * sizeof(float), hipMemcpyHostToDevice));
    }

    // Initialize NCCL
    CHECK_NCCL(ncclCommInitAll(comms.data(), numDevices, devices.data()));

    // benchmark time to all gather START
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Perform All-Gather operation
    for (int i = 0; i < numDevices; i++) {
        CHECK_CUDA(hipSetDevice(i));
        CHECK_NCCL(ncclAllGather(d_sendBuf[i], d_recvBuf[i], dataSize, ncclFloat, comms[i], streams[i]));
    }

    // Synchronize
    for (int i = 0; i < numDevices; i++) {
        CHECK_CUDA(hipSetDevice(i));
        CHECK_CUDA(hipStreamSynchronize(streams[i]));
    }

    // benchmark time to all gather STOP
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "All-Gather time: " << milliseconds << " ms" << std::endl;

    // Copy data back to host and print results
    for (int i = 0; i < numDevices; i++) {
        CHECK_CUDA(hipSetDevice(i));

        std::vector<float> h_recvBuf(totalDataSize);
        CHECK_CUDA(hipMemcpy(h_recvBuf.data(), d_recvBuf[i], totalDataSize * sizeof(float), hipMemcpyDeviceToHost));

        std::cout << "GPU " << i << " received data: ";
        for (float val : h_recvBuf) {
            std::cout << val << " ";
        }
        std::cout << std::endl;
    }

    // Cleanup
    for (int i = 0; i < numDevices; i++) {
        CHECK_CUDA(hipFree(d_sendBuf[i]));
        CHECK_CUDA(hipFree(d_recvBuf[i]));
        CHECK_CUDA(hipStreamDestroy(streams[i]));
        CHECK_NCCL(ncclCommDestroy(comms[i]));
    }
}

int main() {
    all_gather_example();
    return 0;
}
